#include "hip/hip_runtime.h"
#include "math.h"
#include "../glm/glm.hpp"
#include "../glm/vec3.hpp"

#include "core.h"
#include "../basetypes.h"
#include "../SceneObject.h"

using glm::vec3;

__global__
void add(int n, float *x) {
}

namespace core {
    vec3 reflect(vec3 d, vec3 n) {
        return d - (n * glm::dot(d, n) * 2.0f);
    }

    vec3 refract(vec3 d, vec3 n, float ior) {
        float cosi = glm::dot(d, n);
        float etai = 1.0f;
        float etat = ior;

        if (cosi > 0.0f) {
            float t = etai;
            etai = etat;
            etat = t;
            n = -n;
        } else {
            cosi = -cosi;
        }

        float eta = etai / etat;
        float k = 1.0f - eta * eta * (1.0f - cosi * cosi);

        if (k < 0.0f) {
            return vec3(0.0f);
        } else {
            return d * eta + n * (eta * cosi - sqrt(k));
        }
    }

    float fresnel(vec3 d, vec3 n, float ior) {
        float cosi = glm::dot(d, n);
        float etai = 1.0f;
        float etat = ior;

        if (cosi > 0.0f) {
            float t = etai;
            etai = etat;
            etat = t;
        }

        float sint = etai / etat * sqrt(max(1.0f - cosi * cosi, 0.0f));

        if (sint >= 1.0f) {
            return 1.0f;
        } else {
            float cost = sqrt(max(1.0f - sint * sint, 0.0f));
            cosi = abs(cosi);
            float rs = (etat * cosi - etai * cost) / (etat * cosi + etai * cost);
            float rp = (etai * cosi - etat * cost) / (etai * cosi + etat * cost);
            return (rs * rs + rp * rp) / 2.0f;
        }
    }

    Intersection check_intersection(vec3 origin, vec3 dir, SceneObject *objects) {
    }

    // TODO Intersection

    vec3 cast_ray(vec3 pos, vec3 dir, Options options) {
        vec3 color(0.0f);
        vec3 throughput(1.0f);

        for (int i = 0; i < options.max_depth; ++i) {
            Intersection inter;
        }

        return color;
    }

    int test(int n) {
        return n;
    }
}
